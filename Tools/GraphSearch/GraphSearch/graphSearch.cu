#include "hip/hip_runtime.h"
#define BLOCK_SIZE 256
#include "graphNode.h"
#include <cstdlib>
#include <math.h>

__device__ int return_thread_index()
{
	int thread_idx_in_block = threadIdx.x + (threadIdx.y * blockDim.x);
	int block_idx_in_grid = blockIdx.x + (blockIdx.y * gridDim.x);
	return thread_idx_in_block + (block_idx_in_grid * blockDim.x);
}

__global__ void fillPath(graphNode* graph, int sourceIndex, int totalVertex)
{
	int threadIndex = return_thread_index();
	if (threadIndex != sourceIndex && threadIndex < totalVertex)
	{
		int vertex_location = (threadIndex > sourceIndex) ? graph[threadIndex].idx - 1 : graph[threadIndex].idx;
		int current_position = 0;
		int vertexIdx = graph[threadIndex].idx;
		while (vertexIdx != sourceIndex && graph[vertexIdx].val != -1)
		{  
			graph[sourceIndex].paths[vertex_location][current_position] = vertexIdx;
			vertexIdx = graph[vertexIdx].predIdx;
			current_position++;
		}
	}
}

//graph_dev1 -> read
//graph_dev2 -> write
__global__ void graphSearch(graphNode* graph_dev1, graphNode* graph_dev2, int totalVertex, int sourceIndex, int* flag_dev)
{
	int threadIndex = return_thread_index();

	if (threadIndex < totalVertex)
	{
		graphNode vertex = graph_dev1[threadIndex];
		
		if (vertex.val >= 0)
		{
			for (int uidIndex = 0; uidIndex < totalVertex; uidIndex++)
			{
				int neighborIdx = vertex.connectedNodesIdx[uidIndex];
				if (neighborIdx >= 0 && neighborIdx != sourceIndex)
				{
					int neighbor_predIdx = graph_dev1[neighborIdx].predIdx;
					if (neighbor_predIdx == -1) {
						graph_dev2[neighborIdx].val = vertex.val + 1;
						graph_dev2[neighborIdx].predIdx = vertex.idx;
						*flag_dev = 1;
					} else {
						graph_dev2[neighborIdx].val = graph_dev1[neighbor_predIdx].val + 1;
						graph_dev2[neighborIdx].predIdx = neighbor_predIdx;
					}
				}
			}
		}

	}
}


void init_values_for_search(graphNode* graph, int vertexIndex, int totalVertex)
{
	for(int i = 0; i < totalVertex; i++)
	{
		graph[i].val = (vertexIndex != i) ? -1 : 0;
	}
}

extern "C" graphNode* searchGraph(graphNode* graph, int totalVertex)
{
	int blockLength = (int)sqrt((double)BLOCK_SIZE); 
	int gridLength = (int)ceil((double)totalVertex / (double)BLOCK_SIZE);

	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, gridLength, 1);

	graphNode *graph_dev1;
	graphNode *graph_dev2;
	graphNode *graph_out;
	graph_out = (graphNode*)malloc(totalVertex*sizeof(graphNode));
	hipMalloc((void**)&graph_dev1, totalVertex*sizeof(graphNode));
	hipMalloc((void**)&graph_dev2, totalVertex*sizeof(graphNode));

	int *flag, *flag_dev;
	hipMalloc((void**)&flag_dev, sizeof(int));
	flag = (int*)malloc(sizeof(int));

	for (int vertexIndex = 0; vertexIndex < totalVertex; ++vertexIndex)
	{

		init_values_for_search(graph, vertexIndex, totalVertex);

		hipMemcpy(graph_dev1, graph, totalVertex*sizeof(graphNode), hipMemcpyHostToDevice);
		hipMemcpy(graph_dev2, graph, totalVertex*sizeof(graphNode), hipMemcpyHostToDevice);
		

		do {
			*flag = 0;
			hipMemcpy(flag_dev, flag, sizeof(int), hipMemcpyHostToDevice);
			
			graphSearch<<<blocks, threads>>>(graph_dev1, graph_dev2, totalVertex, vertexIndex, flag_dev);


			hipMemcpy(flag, flag_dev, sizeof(int), hipMemcpyDeviceToHost);
			
			graphNode* tmp = graph_dev1;
			graph_dev1 = graph_dev2;
			graph_dev2 = tmp;

		} while (*flag == 1);

		fillPath<<<blocks, threads>>>(graph_dev1, vertexIndex, totalVertex);
		hipMemcpy(&graph_out[vertexIndex], &graph_dev1[vertexIndex], sizeof(graphNode), hipMemcpyDeviceToHost);
	}

	return graph_out;
}